#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <tetmesh.h>

using namespace std;

#define MAX(a,b) std::max(a,b)

#define MIN(a,b) std::min(a,b)

void TetMesh::init(float* pointlist, int numpoint, int*trilist, int numtri, int* tetlist, int numtet, int numattr, float* attrlist, bool verbose)
{

  vertices.resize(numpoint);
  tets.resize(numtet);
  if (verbose)
    printf("number of verts is: %d\n", numpoint);
  if (verbose)
    printf("number of tets is:  %d\n", numtet);
  for(int i = 0; i < numpoint; i++)
  {
    vertices[i][0] = pointlist[3 * i + 0];
    vertices[i][1] = pointlist[3 * i + 1];
    vertices[i][2] = pointlist[3 * i + 2];
  }

  //check the index start from 0 or 1
  int minidx = 1000000000;
  for(int i = 0; i < numtet * 4; i++)
  {
    minidx = MIN(minidx, tetlist[i]);

  }

  if(minidx == 0)
  {
    for(int i = 0; i < numtet; i++)
    {

      tets[i][0] = tetlist[4 * i + 0];
      tets[i][1] = tetlist[4 * i + 1];
      tets[i][2] = tetlist[4 * i + 2];
      tets[i][3] = tetlist[4 * i + 3];
    }
  }
  else if(minidx == 1)
  {
    for(int i = 0; i < numtet; i++)
    {
      tets[i][0] = tetlist[4 * i + 0] - 1; // -1 because the oringal index is from 1 and change it to 0
      tets[i][1] = tetlist[4 * i + 1] - 1;
      tets[i][2] = tetlist[4 * i + 2] - 1;
      tets[i][3] = tetlist[4 * i + 3] - 1;
    }
  }
  else
  {
    printf("error!!! index not start from 0 or 1!!\n");
  }

  if(numattr > 0)
  {
    for(int i = 0; i < numtet; i++)
    {
      int mat = (int)attrlist[i];
      switch(mat)
      {
      case 1:
        tets[i].speedInv = 1.0;
        break;
      case 2:
        tets[i].speedInv = 1.52; // refractive index of glass 1.52
        break;
      case 3:
        tets[i].speedInv = 1.0;
        break;
      }
    }
  }
  else
  {
    for(int i = 0; i < numtet; i++)
      tets[i].speedInv = 1.0;
  }
}
// Find the direct neighbors of each vertex

void TetMesh::need_neighbors(bool verbose)
{
  if(!neighbors.empty())
    return;

  if (verbose)
    cout << "Finding vertex neighbors... " << endl;
  int nv = vertices.size(), nt = tets.size();

  neighbors.resize(nv);
  //for (int i = 0; i < nv; i++)
  //  neighbors[i].reserve(numneighbors[i]+2); // Slop for boundaries

  for(int i = 0; i < nt; i++)
  {
    for(int j = 0; j < 4; j++)
    {
      std::vector<int> &me = neighbors[tets[i][j]];
      int n1 = tets[i][(j + 1) % 4];
      int n2 = tets[i][(j + 2) % 4];
      int n3 = tets[i][(j + 3) % 4];
      if(find(me.begin(), me.end(), n1) == me.end())
        me.push_back(n1);
      if(find(me.begin(), me.end(), n2) == me.end())
        me.push_back(n2);
      if(find(me.begin(), me.end(), n3) == me.end())
        me.push_back(n3);
    }
  }

  if (verbose)
    cout << "Done.\n" << endl;
}

// Find the tets touching each vertex

void TetMesh::need_adjacenttets(bool verbose)
{
  if(!adjacenttets.empty())
    return;

  if (verbose)
    std::cout << "Finding adjacenttets... " << std::endl;
  int nv = vertices.size(), nt = tets.size();

  adjacenttets.resize(vertices.size());

  for(int i = 0; i < nt; i++)
  {
    for(int j = 0; j < 4; j++)
      adjacenttets[tets[i][j]].push_back(i);
  }

  int maxNumAjTets = 0;
  for(int i = 0; i < nv; i++)
  {
    maxNumAjTets = MAX(maxNumAjTets, static_cast<int>(adjacenttets[i].size()));

  }

  if (verbose)
    printf("Max number of adjacent tet is: %d\n", maxNumAjTets);

  if (verbose)
    std::cout << "Done.\n" << std::endl;
}

bool TetMesh::IsNonObtuse(int v, Tet t)
{
  int D = t.indexof(v);
  int A = (D + 1) % 4;
  int B = (D + 2) % 4;
  int C = (D + 3) % 4;

  point P1 = vertices[t[A]];
  point P2 = vertices[t[B]];
  point P3 = vertices[t[C]];
  point P4 = vertices[t[D]];

  point a = P1 - P4;
  point b = P2 - P4;
  point c = P3 - P4;

  LevelsetValueType det = abs((a ^ (b % c)));

  LevelsetValueType al = len(a);
  LevelsetValueType bl = len(b);
  LevelsetValueType cl = len(c);

  LevelsetValueType div = al * bl * cl + (a ^ b) * cl + (a ^ c) * bl + (b ^ c) * al;
  LevelsetValueType at = atan2(det, div);
  if(at < 0) at += M_PI; // If det>0 && div<0 atan2 returns < 0, so add pi.
  LevelsetValueType omega = 2.0f * at;

  return omega < M_PI / 2.0;
}

void TetMesh::SplitFace(vector<Tet> &acTets, int v, Tet ct, int nfAdj)
{
  // get all the four vertices in order
  /* v1         v4
     +-------+
     \     . \
     \   .   \
     \ .     \
     +-------+
     v2         v3 */

  need_neighbors();
  int iV = ct.indexof(v); // get index of v in terms of cf
  int v1 = v;
  int v2 = ct[(iV + 1) % 4];
  int v3 = ct[(iV + 2) % 4];
  int v4 = ct[(iV + 3) % 4];
  iV = tets[nfAdj].indexof(v2); // get index of v in terms of adjacent face

  int v5;
  for(int i = 0; i < 4; i++)
  {
    if(tets[nfAdj][i] != v2 && tets[nfAdj][i] != v3 && tets[nfAdj][i] != v4)
      v5 = tets[nfAdj][i];

  }
  neighbors[v1].push_back(v5);
  //Tet af = tets[nfAdj];

  // create faces (v1,v3,v4) and (v1,v2,v3), check angle at v1
  Tet t1(v1, v2, v3, v5);
  Tet t2(v1, v3, v4, v5);
  Tet t3(v1, v2, v4, v5);



  if(IsNonObtuse(v, t1))
  {
    acTets.push_back(t1);
  }
  else
  {
    int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v4)];
    if(nfAdj_new > -1)
    {
      SplitFace(acTets, v, t1, nfAdj_new);

    }
    else
      printf("NO cross edge!!! Maybe a hole!!\n");
    //SplitFace(acFaces,v,f1,nfAdj_new, currentVert);
  }

  if(IsNonObtuse(v, t2))
  {
    acTets.push_back(t2);
  }
  else
  {
    int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v2)];
    if(nfAdj_new > -1)
    {
      SplitFace(acTets, v, t2, nfAdj_new/*,currentVert*/);
    }
    else
      printf("NO cross edge!!! Maybe a hole!!\n");
    //SplitFace(acFaces,v,f2,nfAdj_new,currentVert);
  }

  if(IsNonObtuse(v, t3))
  {
    acTets.push_back(t3);
  }
  else
  {
    int nfAdj_new = across_face[nfAdj][tets[nfAdj].indexof(v3)];
    if(nfAdj_new > -1)
    {
      SplitFace(acTets, v, t3, nfAdj_new/*,currentVert*/);
    }
    else
      printf("NO cross edge!!! Maybe a hole!!\n");
  }
}

void TetMesh::need_across_face()
{
  if(!across_face.empty())
    return;
  need_adjacenttets();

  printf("Finding across-face maps... ");

  int nt = tets.size();
  across_face.resize(nt, Tet(-1, -1, -1, -1));

  for(int i = 0; i < nt; i++)
  {
    for(int j = 0; j < 4; j++)
    {
      if(across_face[i][j] != -1)
        continue;
      int v1 = tets[i][(j + 1) % 4];
      int v2 = tets[i][(j + 2) % 4];
      int v3 = tets[i][(j + 3) % 4];
      const vector<int> &a1 = adjacenttets[v1];
      const vector<int> &a2 = adjacenttets[v2];
      const vector<int> &a3 = adjacenttets[v3];
      for(int k1 = 0; k1 < a1.size(); k1++)
      {
        int other = a1[k1];
        if(other == i)
          continue;
        vector<int>::const_iterator it =
          find(a2.begin(), a2.end(), other);

        vector<int>::const_iterator it2 =
          find(a3.begin(), a3.end(), other);

        if(it == a2.end() || it2 == a3.end())
          continue;

        across_face[i][j] = other;
        break;



      }
    }
  }
  printf("Done.\n");
}

vector<TetMesh::Tet> TetMesh::GetOneRing(int v)
{
  // make sure we have the across-edge map
  if(across_face.empty())
    need_across_face();

  // variables required
  vector<Tet> oneRingTets;
  vector<Tet> t_tets;

  // get adjacent faces
  int naf = adjacenttets[v].size();

  if(!naf)
  {
    std::cout << "vertex " << v << " has 0 adjacent faces..." << std::endl;
  }
  else
  {
    for(int af = 0; af < naf; af++)
    {
      Tet ct = this->tets[adjacenttets[v][af]];

      t_tets.clear();
      if(IsNonObtuse(v, ct))// check angle: if non-obtuse, return existing face
      {
        t_tets.push_back(ct);
      }
      else
      {
        int nfae = this->across_face[adjacenttets[v][af]][ct.indexof(v)];
        if(nfae > -1)
        {
          SplitFace(t_tets, v, ct, nfae/*,currentVert*/); // if obtuse, split face till we get all acute angles
        }
        else
          printf("NO cross edge!!! Maybe a hole!!\n");
        //SplitFace(t_faces,v,cf,nfae,currentVert);// if obtuse, split face till we get all acute angles
      }

      for(int tf = 0; tf < t_tets.size(); tf++)
      {
        oneRingTets.push_back(t_tets[tf]);
      }
    }
  }
  return oneRingTets;
}

void TetMesh::need_oneringtets()
{

  if(vertOneringTets.empty())
  {
    vertOneringTets.resize(vertices.size());
    for(int i = 0; i < vertices.size(); i++)
    {
      vertOneringTets[i] = GetOneRing(i);
    }
  }
}

void TetMesh::reorient()
{
  int ne = tets.size();
  for(int i = 0; i < ne; i++)
  {
    Tet& t = tets[i];
    point A = vertices[t[0]];
    point B = vertices[t[1]];
    point C = vertices[t[2]];
    point D = vertices[t[3]];
    point AB = B - A;
    point AC = C - A;
    point AD = D - A;

    LevelsetValueType tmp = ((AB)CROSS(AC)) DOT(AD);
    if(tmp < 0)
    {
      int tmpidx = t[1];
      t[1] = t[2];
      t[2] = tmpidx;
    }
  }
}

void TetMesh::rescale(LevelsetValueType size)
{

  LevelsetValueType minx = LARGENUM;
  LevelsetValueType miny = LARGENUM;
  LevelsetValueType minz = LARGENUM;
  LevelsetValueType maxx = -LARGENUM;
  LevelsetValueType maxy = -LARGENUM;
  LevelsetValueType maxz = -LARGENUM;
  for(int v = 0; v < vertices.size(); v++)
  {
    LevelsetValueType x = vertices[v][0];
    LevelsetValueType y = vertices[v][1];
    LevelsetValueType z = vertices[v][2];
    if(x < minx)
      minx = x;
    if(y < miny)
      miny = y;
    if(z < minz)
      minz = z;

    if(x > maxx)
      maxx = x;
    if(y > maxy)
      maxy = y;
    if(z > maxz)
      maxz = z;
  }
  for(int v = 0; v < vertices.size(); v++)
  {
    vertices[v][0] -= minx;
    vertices[v][1] -= miny;
    vertices[v][2] -= minz;

    vertices[v][0] = vertices[v][0] / (maxx - minx) * size;
    vertices[v][1] = vertices[v][1] / (maxy - miny) * size;
    vertices[v][2] = vertices[v][2] / (maxz - minz) * size;
  }
}

