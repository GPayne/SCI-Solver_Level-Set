#include "hip/hip_runtime.h"
#include <meshFIM.h>
#include <tetmesh.h>
#include <Vec.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <meshFIM_kernels.h>
#include <mycutil.h>
#include <cusp/detail/format_utils.h>
#include <cusp/print.h>
#include <thrust/functional.h>

extern "C"
{
#include <metis.h>
}

void meshFIM::writeFLD()
{
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  FILE* matfile;
  FILE* fldfile;
  matfile = fopen("result.txt", "w+");
  fldfile = fopen("result.fld", "w+");
  fprintf(fldfile, "SCI\nASC\n2\n{@1 {GenericField<TetVolMesh<TetLinearLgn<Point>>,ConstantBasis<float>,vector<float>> 3 {Field 3 {PropertyManager 2 0 }\n}\n{@2 {TetVolMesh<TetLinearLgn<Point>> 4 {Mesh 2 {PropertyManager 2 0 }\n}\n");
  fprintf(fldfile, "{STLVector 2 %d ", nv);
  for(int i = 0; i < nv; i++)
  {
    fprintf(fldfile, "{%.12f %.12f %.12f}", m_meshPtr->vertices[i][0], m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
  }
  fprintf(fldfile, "}\n{STLIndexVector 1 %d 8 ", nt * 4);
  for(int i = 0; i < nt; i++)
  {
    fprintf(fldfile, "%d %d %d %d ", m_meshPtr->tets[i][0], m_meshPtr->tets[i][1], m_meshPtr->tets[i][2], m_meshPtr->tets[i][3]);
  }
  fprintf(fldfile, "}\n");
  fprintf(fldfile, "{TetLinearLgn<Point>  1 }\n}\n}{ConstantBasis<float>  1 }\n");
  fprintf(fldfile, "{STLVector 2 %d ", nt);
  for(int i = 0; i < nt; i++)
  {
    fprintf(fldfile, " 0");
  }

  fprintf(fldfile, "}\n}\n}");
  for(int i = 0; i < nv; i++)
  {
    fprintf(matfile, "%.12f\n", m_meshPtr->vertT[i]);
  }
  fclose(matfile);
  fclose(fldfile);
}

void meshFIM::writeVTK(std::vector < std::vector <LevelsetValueType> > values)
{
  FILE* vtkfile;
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  vtkfile = fopen("result.vtk", "w+");
  fprintf(vtkfile, "# vtk DataFile Version 3.0\nvtk output\nASCII\nDATASET UNSTRUCTURED_GRID\n");
  fprintf(vtkfile, "POINTS %d float\n", nv);
  for(int i = 0; i < nv; i++)
  {
    fprintf(vtkfile, "%.12f %.12f %.12f\n", m_meshPtr->vertices[i][0], m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
  }
  fprintf(vtkfile, "CELLS %d %d\n", nt, nt * 5);
  for(int i = 0; i < nt; i++)
  {
    fprintf(vtkfile, "4 %d %d %d %d\n", m_meshPtr->tets[i][0], m_meshPtr->tets[i][1], m_meshPtr->tets[i][2], m_meshPtr->tets[i][3]);
  }

  fprintf(vtkfile, "CELL_TYPES %d\n", nt);
  for(int i = 0; i < nt; i++)
  {
    fprintf(vtkfile, "10\n");
  }
  fprintf(vtkfile, "POINT_DATA %d\nSCALARS traveltime float %lu\nLOOKUP_TABLE default\n",
      nv, values.size());
  for (int j = 0; j < values[0].size(); j++) {
    for (size_t i = 0; i < values.size(); i++) {
      fprintf(vtkfile, "%.12f ", values[i][j]);
    }
    fprintf(vtkfile,"\n");
  }
  fclose(vtkfile);
}

void meshFIM::updateT_single_stage_d(LevelsetValueType timestep, int niter, IdxVector_d& narrowband, int num_narrowband)
{
  int nn = m_meshPtr->vertices.size();
  int nblocks = num_narrowband;
  int nthreads = largest_ele_part;
  thrust::fill(vertT_out.begin(), vertT_out.end(), 0.0);
  int shared_size = sizeof(LevelsetValueType)* 4 * largest_ele_part + sizeof(short)*largest_vert_part*m_largest_num_inside_mem;
  cudaSafeCall((kernel_updateT_single_stage << <nblocks, nthreads, shared_size >> >(timestep, CAST(narrowband), largest_ele_part, largest_vert_part, full_num_ele,
          CAST(m_ele_after_permute_d), CAST(m_ele_offsets_d), CAST(m_cadv_local_d),
          nn, CAST(m_vert_offsets_d), CAST(m_vert_after_permute_d), CAST(m_vertT_after_permute_d),
          CAST(m_ele_local_coords_d), m_largest_num_inside_mem, CAST(m_mem_locations), CAST(m_mem_location_offsets),
          CAST(vertT_out))));

  nthreads = largest_vert_part;
  cudaSafeCall((CopyOutBack_levelset << <nblocks, nthreads >> >(CAST(narrowband),
          CAST(m_vert_offsets_d), CAST(m_vertT_after_permute_d), CAST(vertT_out))));
}

//Single stage update

void meshFIM::updateT_single_stage(LevelsetValueType timestep, int nside, int niter, vector<int>& narrowband)
{
  vec3 sigma(1.0, 0.0, 1.0);
  LevelsetValueType epsilon = 1.0;
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  vector<LevelsetValueType> values(4);
  vector<LevelsetValueType> up(nv, 0.0);
  vector<LevelsetValueType> down(nv, 0.0);
  vector<vec3> node_grad_phi_up(nv, vec3(0.0, 0.0, 0.0));
  vector<LevelsetValueType> node_grad_phi_down(nv, 0.0);
  vector<LevelsetValueType> curv_up(nv, 0.0);


  for(int bandidx = 0; bandidx < narrowband.size(); bandidx++)
  {
    int tidx = narrowband[bandidx];
    for(int j = 0; j < 4; j++)
    {
      values[j] = m_meshPtr->vertT[m_meshPtr->tets[tidx][j]];
    }
    //compute ni normals
    vector<vec3> nodes(4);
    nodes[0] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][0]];
    nodes[1] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][1]];
    nodes[2] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][2]];
    nodes[3] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][3]];
    vec3 v31 = nodes[1] - nodes[3];
    vec3 v32 = nodes[2] - nodes[3];
    vec3 v30 = nodes[0] - nodes[3];
    vec3 crossproduct = v31 CROSS v32;
    LevelsetValueType dotproduct = crossproduct DOT v30;
    LevelsetValueType volume = fabs(dotproduct) / 6.0;

    //compute inverse of 4 by 4 matrix
    LevelsetValueType a11 = nodes[0][0], a12 = nodes[0][1], a13 = nodes[0][2], a14 = 1.0;
    LevelsetValueType a21 = nodes[1][0], a22 = nodes[1][1], a23 = nodes[1][2], a24 = 1.0;
    LevelsetValueType a31 = nodes[2][0], a32 = nodes[2][1], a33 = nodes[2][2], a34 = 1.0;
    LevelsetValueType a41 = nodes[3][0], a42 = nodes[3][1], a43 = nodes[3][2], a44 = 1.0;

    LevelsetValueType det =
      a11 * a22 * a33 * a44 + a11 * a23 * a34 * a42 + a11 * a24 * a32 * a43
      + a12 * a21 * a34 * a43 + a12 * a23 * a31 * a44 + a12 * a24 * a33 * a41
      + a13 * a21 * a32 * a44 + a13 * a22 * a34 * a41 + a13 * a24 * a31 * a42
      + a14 * a21 * a33 * a42 + a14 * a22 * a31 * a43 + a14 * a23 * a32 * a41
      - a11 * a22 * a34 * a43 - a11 * a23 * a32 * a44 - a11 * a24 * a33 * a42
      - a12 * a21 * a33 * a44 - a12 * a23 * a34 * a41 - a12 * a24 * a31 * a43
      - a13 * a21 * a34 * a42 - a13 * a22 * a31 * a44 - a13 * a24 * a32 * a41
      - a14 * a21 * a32 * a43 - a14 * a22 * a33 * a41 - a14 * a23 * a31 * a42;

    LevelsetValueType b11 = a22 * a33 * a44 + a23 * a34 * a42 + a24 * a32 * a43 - a22 * a34 * a43 - a23 * a32 * a44 - a24 * a33 * a42;
    LevelsetValueType b12 = a12 * a34 * a43 + a13 * a32 * a44 + a14 * a33 * a42 - a12 * a33 * a44 - a13 * a34 * a42 - a14 * a32 * a43;
    LevelsetValueType b13 = a12 * a23 * a44 + a13 * a24 * a42 + a14 * a22 * a43 - a12 * a24 * a43 - a13 * a22 * a44 - a14 * a23 * a42;
    LevelsetValueType b14 = a12 * a24 * a33 + a13 * a22 * a34 + a14 * a23 * a32 - a12 * a23 * a34 - a13 * a24 * a32 - a14 * a22 * a33;

    LevelsetValueType b21 = a21 * a34 * a43 + a23 * a31 * a44 + a24 * a33 * a41 - a21 * a33 * a44 - a23 * a34 * a41 - a24 * a31 * a43;
    LevelsetValueType b22 = a11 * a33 * a44 + a13 * a34 * a41 + a14 * a31 * a43 - a11 * a34 * a43 - a13 * a31 * a44 - a14 * a33 * a41;
    LevelsetValueType b23 = a11 * a24 * a43 + a13 * a21 * a44 + a14 * a23 * a41 - a11 * a23 * a44 - a13 * a24 * a41 - a14 * a21 * a43;
    LevelsetValueType b24 = a11 * a23 * a34 + a13 * a24 * a31 + a14 * a21 * a33 - a11 * a24 * a33 - a13 * a21 * a34 - a14 * a23 * a31;


    LevelsetValueType b31 = a21 * a32 * a44 + a22 * a34 * a41 + a24 * a31 * a42 - a21 * a34 * a42 - a22 * a31 * a44 - a24 * a32 * a41;
    LevelsetValueType b32 = a11 * a34 * a42 + a12 * a31 * a44 + a14 * a32 * a41 - a11 * a32 * a44 - a12 * a34 * a41 - a14 * a31 * a42;
    LevelsetValueType b33 = a11 * a22 * a44 + a12 * a24 * a41 + a14 * a21 * a42 - a11 * a24 * a42 - a12 * a21 * a44 - a14 * a22 * a41;
    LevelsetValueType b34 = a11 * a24 * a32 + a12 * a21 * a34 + a14 * a22 * a31 - a11 * a22 * a34 - a12 * a24 * a31 - a14 * a21 * a32;

    LevelsetValueType b41 = a21 * a33 * a42 + a22 * a31 * a43 + a23 * a32 * a41 - a21 * a32 * a43 - a22 * a33 * a41 - a23 * a31 * a42;
    LevelsetValueType b42 = a11 * a32 * a43 + a12 * a33 * a41 + a13 * a31 * a42 - a11 * a33 * a42 - a12 * a31 * a43 - a13 * a32 * a41;
    LevelsetValueType b43 = a11 * a23 * a42 + a12 * a21 * a43 + a13 * a22 * a41 - a11 * a22 * a43 - a12 * a23 * a41 - a13 * a21 * a42;
    LevelsetValueType b44 = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 - a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

    vector<vec4> Arows(4);
    Arows[0] = vec4(b11 / det, b12 / det, b13 / det, b14 / det);
    Arows[1] = vec4(b21 / det, b22 / det, b23 / det, b24 / det);
    Arows[2] = vec4(b31 / det, b32 / det, b33 / det, b34 / det);
    Arows[3] = vec4(b41 / det, b42 / det, b43 / det, b44 / det);

    vector<vec3> nablaN(4);
    for(int i = 0; i < 4; i++)
    {
      vec4 RHS(0.0, 0.0, 0.0, 0.0);
      RHS[i] = 1.0;
      nablaN[i][0] = Arows[0] DOT RHS;
      nablaN[i][1] = Arows[1] DOT RHS;
      nablaN[i][2] = Arows[2] DOT RHS;
    }

    //compuate grad of Phi
    vec3 nablaPhi(0.0, 0.0, 0.0);
    for(int i = 0; i < 4; i++)
    {
      nablaPhi[0] += nablaN[i][0] * values[i];
      nablaPhi[1] += nablaN[i][1] * values[i];
      nablaPhi[2] += nablaN[i][2] * values[i];
    }
    LevelsetValueType abs_nabla_phi = len(nablaPhi);

    //compute K and Kplus and Kminus
    vector<LevelsetValueType> Kplus(4);
    vector<LevelsetValueType> Kminus(4);
    vector<LevelsetValueType> K(4);
    LevelsetValueType Hintegral = 0.0;
    LevelsetValueType beta = 0;
    for(int i = 0; i < 4; i++)
    {
      K[i] = volume * (sigma DOT nablaN[i]); // for H(\nabla u) = sigma DOT \nabla u
      Hintegral += K[i] * values[i];
      Kplus[i] = fmax(K[i], (LevelsetValueType)0.0);
      Kminus[i] = fmin(K[i], (LevelsetValueType)0.0);
      beta += Kminus[i];
    }
    beta = 1.0 / beta;

    if(fabs(Hintegral) > 1e-16)
    {
      vector<LevelsetValueType> delta(4);
      for(int i = 0; i < 4; i++)
      {
        delta[i] = Kplus[i] * beta * (Kminus[0] * (values[i] - values[0]) + Kminus[1] * (values[i] - values[1]) + Kminus[2] * (values[i] - values[2]) + Kminus[3] * (values[i] - values[3]));
      }

      vector<LevelsetValueType> alpha(4);
      for(int i = 0; i < 4; i++)
      {
        alpha[i] = delta[i] / Hintegral;
      }

      LevelsetValueType theta = 0;
      for(int i = 0; i < 4; i++)
      {
        theta += fmax((LevelsetValueType)0.0, alpha[i]);
      }

      vector<LevelsetValueType> alphatuda(4);
      for(int i = 0; i < 4; i++)
      {
        alphatuda[i] = fmax(alpha[i], (LevelsetValueType)0.0) / theta;
      }

      for(int i = 0; i < 4; i++)
      {
        up[m_meshPtr->tets[tidx][i]] += alphatuda[i] * Hintegral;
        down[m_meshPtr->tets[tidx][i]] += alphatuda[i] * volume;
        node_grad_phi_up[m_meshPtr->tets[tidx][i]] += volume* nablaPhi;
        node_grad_phi_down[m_meshPtr->tets[tidx][i]] += volume;
        curv_up[m_meshPtr->tets[tidx][i]] += volume * ((nablaN[i] DOT nablaN[i]) / abs_nabla_phi * values[i] +
            (nablaN[i] DOT nablaN[(i + 1) % 4]) / abs_nabla_phi * values[(i + 1) % 4] +
            (nablaN[i] DOT nablaN[(i + 2) % 4]) / abs_nabla_phi * values[(i + 2) % 4] +
            (nablaN[i] DOT nablaN[(i + 3) % 4]) / abs_nabla_phi * values[(i + 3) % 4]);
      }
    }
  }

  for(int vidx = 0; vidx < nv; vidx++)
  {
    LevelsetValueType eikonal = up[vidx] / down[vidx];
    LevelsetValueType curvature = curv_up[vidx] / node_grad_phi_down[vidx];
    LevelsetValueType node_eikonal = len(node_grad_phi_up[vidx]) / node_grad_phi_down[vidx];
    if(fabs(down[vidx]) > 1e-16)
    {
      m_meshPtr->vertT[vidx] -= epsilon * node_eikonal * curvature * timestep;
    }
  }
}

void meshFIM::GraphPartition_Square(int squareLength, int squareWidth, int squareHeight, int blockLength, int blockWidth, int blockHeight, bool verbose)
{
  int nn = m_meshPtr->vertices.size();
  int numBlockLength = ceil((LevelsetValueType)squareLength / blockLength);
  int numBlockWidth = ceil((LevelsetValueType)squareWidth / blockWidth);
  int numBlockHeight = ceil((LevelsetValueType)squareHeight / blockHeight);
  int numBlock = numBlockLength * numBlockWidth*numBlockHeight;
  npart_h = IdxVector_h(nn);
  nparts = numBlock;

  int edgeCount = 0;
  for(int vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();
  }

  m_largest_num_inside_mem = 0;
  for(int i = 0; i < nn; i++)
  {
    if(m_meshPtr->adjacenttets[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacenttets[i].size();
  }
  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);

  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for(int i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for(int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;

  for(int k = 0; k < squareHeight; k++)
    for(int i = 0; i < squareWidth; i++)
      for(int j = 0; j < squareLength; j++)
      {
        int index = k * squareLength * squareWidth + i * squareLength + j;
        int k2 = k;
        int i2 = i;
        int j2 = j;
        npart_h[index] = (k2 / blockHeight) * numBlockLength *
          numBlockWidth + (i2 / blockWidth) * numBlockLength + (j2 / blockLength);
      }

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  if (verbose) {
    std::cout << npart_h.size() << std::endl;
    std::cout << part_sizes.size() << std::endl;
    std::cout << nn << std::endl;
  }
  for(int i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), part_sizes.end(),
      100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), part_sizes.end(),
      -1, thrust::maximum<int>());
  if(verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if(min_part_size == 0)
    if(verbose)
      printf("Min partition size is 0!!\n");
  delete[] xadj;
  delete[] adjncy;
}

void meshFIM::Partition_METIS(int metissize, bool verbose)
{
  int options[10], pnumflag = 0, wgtflag = 0;
  options[0] = 0;
  int edgecut;
  int nn = m_meshPtr->vertices.size();
  npart_h = IdxVector_h(nn);
  nparts = ceil((LevelsetValueType)nn / (LevelsetValueType)metissize);

  // Counting up edges for adjacency:
  int edgeCount = 0;
  for(int vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();
  }

  m_largest_num_inside_mem = 0;
  for(int i = 0; i < nn; i++)
  {
    if(m_meshPtr->adjacenttets[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacenttets[i].size();
  }
  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);


  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for(int i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for(int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;

  METIS_PartGraphKway(&nn, xadj, adjncy, NULL, NULL, &wgtflag, &pnumflag, &nparts, options, &edgecut, thrust::raw_pointer_cast(&npart_h[0]));

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  for(int i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), part_sizes.end(), 100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), part_sizes.end(), -1, thrust::maximum<int>());
  if (verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if(min_part_size == 0)
    if (verbose)
      printf("Min partition size is 0!!\n");
  delete [] xadj;
  delete [] adjncy;
}

void meshFIM::InitPatches(bool verbose)
{
  int ne = m_meshPtr->tets.size();
  int nn = m_meshPtr->vertices.size();
  ele_d = IdxVector_d(4 * ne);
  ele_h = IdxVector_h(4 * ne);
  vert_d = Vector_d(3 * nn);
  m_vert_after_permute_d = Vector_d(3 * nn);
  Vector_h vert_h(3 * nn);
  for(int eidx = 0; eidx < ne; eidx++)
  {
    for(int i = 0; i < 4; i++)
      ele_h[i * ne + eidx] = m_meshPtr->tets[eidx][i]; //interleaved storage
  }
  for(int vidx = 0; vidx < nn; vidx++)
  {
    for(int i = 0; i < 3; i++)
      vert_h[i * nn + vidx] = m_meshPtr->vertices[vidx][i]; //interleaved storage
  }
  ele_d = ele_h;
  vert_d = vert_h;
  m_npart_d = IdxVector_d(npart_h.begin(), npart_h.end());
  m_part_label_d = IdxVector_d(m_npart_d.begin(), m_npart_d.end());
  int nthreads = 256;
  int nblocks = min((int)ceil((LevelsetValueType)ne / nthreads), 65535);
  cudaSafeCall((kernel_compute_ele_npart << <nblocks, nthreads >> >(ne, thrust::raw_pointer_cast(&m_npart_d[0]), thrust::raw_pointer_cast(&ele_d[0]), thrust::raw_pointer_cast(&ele_label_d[0]))));


  full_num_ele = thrust::reduce(ele_label_d.begin(), ele_label_d.end());
  if(verbose)
    printf("full_num_ele = %d\n", full_num_ele);
  ele_offsets_d[0] = 0;
  thrust::inclusive_scan(ele_label_d.begin(), ele_label_d.end(), ele_offsets_d.begin() + 1);
  ele_full_label = IdxVector_d(full_num_ele);
  ele_permute = IdxVector_d(full_num_ele);

  cudaSafeCall((kernel_fill_ele_label << <nblocks, nthreads >> >(ne, thrust::raw_pointer_cast(&ele_permute[0]), thrust::raw_pointer_cast(&ele_offsets_d[0]),
          thrust::raw_pointer_cast(&m_npart_d[0]), thrust::raw_pointer_cast(&ele_d[0]),
          thrust::raw_pointer_cast(&ele_full_label[0]))));

  clock_t starttime, endtime;
  double duration;
  starttime = clock();
  thrust::sort_by_key(ele_full_label.begin(), ele_full_label.end(), ele_permute.begin());
  hipDeviceSynchronize();
  endtime = clock();
  duration = (double)(endtime - starttime) / (double)CLOCKS_PER_SEC;
  if(verbose)
    printf("Sorting time : %.10lf s\n", duration);
  m_ele_offsets_d = IdxVector_d(nparts + 1);
  ones = IdxVector_d(full_num_ele, 1);
  tmp = IdxVector_d(full_num_ele);
  reduce_output = IdxVector_d(full_num_ele);
  thrust::reduce_by_key(ele_full_label.begin(),
      ele_full_label.end(), ones.begin(), tmp.begin(), reduce_output.begin());
  largest_ele_part = thrust::reduce(reduce_output.begin(),
      reduce_output.begin() + nparts, -1, thrust::maximum<int>());
  if(verbose)
    printf("Largest element partition size is: %d\n", largest_ele_part);
  if(largest_ele_part > 1024)
  {
    printf("Error: largest_ele_part > 1024 !!\n");
    exit(0);
  }
  m_ele_offsets_d[0] = 0;
  thrust::inclusive_scan(reduce_output.begin(), reduce_output.begin() + nparts, m_ele_offsets_d.begin() + 1);
}

void meshFIM::InitPatches2()
{
  int ne = m_meshPtr->tets.size();
  int nn = m_meshPtr->vertices.size();
  IdxVector_d vert_permute(nn, 0);
  IdxVector_d vert_ipermute(nn, 0);
  int nthreads = 256;
  int nblocks = min((int)ceil((LevelsetValueType)nn / nthreads), 65535);
  cudaSafeCall((kernel_fill_sequence << <nblocks, nthreads >> >(nn, CAST(vert_permute))));
  thrust::sort_by_key(m_part_label_d.begin(), m_part_label_d.end(), vert_permute.begin());
  nblocks = min((int)ceil((LevelsetValueType)nn / nthreads), 65535);
  cudaSafeCall((kernel_compute_vert_ipermute << <nblocks, nthreads >> >(nn, thrust::raw_pointer_cast(&vert_permute[0]), thrust::raw_pointer_cast(&vert_ipermute[0]))));

  m_vert_permute_d = IdxVector_d(vert_permute);
  m_vert_offsets_d = IdxVector_d(nparts + 1);
  cusp::detail::indices_to_offsets(m_part_label_d, m_vert_offsets_d);

  //permute the vert and ele values
  m_ele_after_permute_d = IdxVector_d(4 * full_num_ele);
  m_vertT_after_permute_d = Vector_d(nn);
  nblocks = min((int)ceil((LevelsetValueType)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_ele_and_vert << <nblocks, nthreads >> >(full_num_ele, ne,
          thrust::raw_pointer_cast(&ele_d[0]), thrust::raw_pointer_cast(&m_ele_after_permute_d[0]), thrust::raw_pointer_cast(&ele_permute[0]),
          nn, thrust::raw_pointer_cast(&vert_d[0]), thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
          thrust::raw_pointer_cast(&m_vertT_d[0]), thrust::raw_pointer_cast(&m_vertT_after_permute_d[0]),
          CAST(vert_permute),
          thrust::raw_pointer_cast(&vert_ipermute[0]))));

  //compute the local coords for each element
  m_ele_local_coords_d = Vector_d(6 * full_num_ele);
  m_cadv_local_d = Vector_d(3 * full_num_ele);
  nthreads = 256;
  nblocks = min((int)ceil((LevelsetValueType)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_compute_local_coords << <nblocks, nthreads >> >(full_num_ele, nn,
          thrust::raw_pointer_cast(&m_ele_after_permute_d[0]), thrust::raw_pointer_cast(&m_ele_offsets_d[0]),
          thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
          thrust::raw_pointer_cast(&m_ele_local_coords_d[0]),
          CAST(m_cadv_global_d), CAST(m_cadv_local_d))));
  //Generate redution list

  m_mem_locations = IdxVector_d(4 * full_num_ele);
  IdxVector_d tmp2 = m_ele_after_permute_d;
  thrust::sequence(m_mem_locations.begin(), m_mem_locations.end(), 0);
  thrust::sort_by_key(tmp2.begin(), tmp2.end(), m_mem_locations.begin());
  m_mem_location_offsets = IdxVector_d(nn + 1);
  cusp::detail::indices_to_offsets(tmp2, m_mem_location_offsets);

}

void meshFIM::GenerateBlockNeighbors()
{

  //Generate block neighbors
  // Declaring temporary vectors:
  adjacencyBlockLabel = IdxVector_d(m_adjncy_d.size(), 0);
  blockMappedAdjacency = IdxVector_d(m_adjncy_d.size(), 0);

  mapAdjacencyToBlock(m_xadj_d, m_adjncy_d, adjacencyBlockLabel, blockMappedAdjacency, m_npart_d);
  // Zip up the block label and block mapped vectors and sort:
  thrust::sort(thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.end(), blockMappedAdjacency.end())));

  // Remove Duplicates and resize:
  int newSize = thrust::unique(thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.end(), blockMappedAdjacency.end()))) -
    thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin()));


  adjacencyBlockLabel.resize(newSize);
  blockMappedAdjacency.resize(newSize);
  getPartIndicesNegStart(adjacencyBlockLabel, m_block_xadj_d);
  m_block_adjncy_d.resize(blockMappedAdjacency.size() - 1);
  thrust::copy(blockMappedAdjacency.begin() + 1, blockMappedAdjacency.end(), m_block_adjncy_d.begin());
}

std::vector <std::vector <LevelsetValueType> > meshFIM::GenerateData(
    char* filename, int nsteps, LevelsetValueType timestep, int inside_niter,
    int nside, int block_size, LevelsetValueType bandwidth, int part_type,
    int metis_size, double domain, int axis, bool verbose)
{
  if (verbose)
    printf("Starting meshFIM::GenerateData\n");
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();

  int squareLength = nside;
  int squareWidth = nside;
  int squareDepth = nside;
  int squareBlockLength = block_size;
  int squareBlockWidth = block_size;
  int squareBlockDepth = block_size;
  //  LevelsetValueType starttime, endtime, duration;
  clock_t starttime, endtime, starttime1, endtime1;
  LevelsetValueType duration, duration1 = 0.0, duration2 = 0.0;

  if(part_type == 1)
    GraphPartition_Square(squareLength, squareWidth, squareDepth, squareBlockLength, squareBlockWidth, squareBlockDepth, verbose);
  else //partition with METIS
  {
    Partition_METIS(metis_size, verbose);
  }
  //Initialize the values
  if(m_meshPtr->vertT.size() == 0)
    m_meshPtr->vertT.resize(nv);

  Vector_h h_vertT(nv);
  for(int i = 0; i < nv; i++)
  {
    vec3 v1 = (vec3)m_meshPtr->vertices[i];
    m_meshPtr->vertT[i] = v1[axis] - domain;
    h_vertT[i] = m_meshPtr->vertT[i];
  }
  m_vertT_d = h_vertT;

  starttime = clock();
  //Init patches
  InitPatches(verbose);
  Vector_h cadv_h(3 * full_num_ele);
  for(int i = 0; i < full_num_ele; i++)
  {
    cadv_h[0 * full_num_ele + i] = axis==0?1.0:0.0;
    cadv_h[1 * full_num_ele + i] = axis==1?1.0:0.0;
    cadv_h[2 * full_num_ele + i] = axis==2?1.0:0.0;
  }
  m_cadv_global_d = Vector_d(cadv_h);
  InitPatches2();
  GenerateBlockNeighbors();
  if (verbose)
    printf("After  preprocessing\n");
  endtime = clock();
  duration = (LevelsetValueType)(endtime - starttime) / CLOCKS_PER_SEC;
  if (verbose)
    printf("pre processing time : %.10lf s\n", duration);

  //Inite redistance
  m_redist = new redistance(m_meshPtr, nparts, m_block_xadj_d, m_block_adjncy_d);

  //////////////////////////update values///////////////////////////////////////////
  IdxVector_d narrowband_d(nparts);
  int num_narrowband = 0;

  std::vector <std::vector <LevelsetValueType> >  ans;

  starttime = clock();
  for(int stepcount = 0; stepcount < nsteps; stepcount++)
  {
    m_redist->FindSeedPoint(narrowband_d, num_narrowband, m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d);

    m_redist->ReInitTsign(m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d);
    starttime1 = clock();
    m_redist->GenerateData(narrowband_d, num_narrowband, bandwidth, stepcount, m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d, verbose);
    hipDeviceSynchronize();
    endtime1 = clock();
    duration1 += endtime1 - starttime1;
    if (num_narrowband == 0) {
      std::cout << "NOTE: Ending at timestep " << stepcount <<
        " due to zero narrow band." << std::endl;
      break;
    }
    starttime1 = clock();
    for(int niter = 0; niter < inside_niter; niter++)
      updateT_single_stage_d(timestep, stepcount, narrowband_d, num_narrowband);

    hipDeviceSynchronize();
    endtime1 = clock();
    duration2 += endtime1 - starttime1;
    ///////////////////done updating/////////////////////////////////////////////////
    int nthreads = 256;
    int nblocks = min((int)ceil((LevelsetValueType)nv / nthreads), 655535);
    cudaSafeCall((kernel_compute_vertT_before_permute << <nblocks, nthreads >> >(nv, CAST(m_vert_permute_d), CAST(m_vertT_after_permute_d), CAST(tmp_vertT_before_permute_d))));
    Vector_h vertT_before_permute_h = tmp_vertT_before_permute_d;
    for(int i = 0; i < nv; i++)
    {
      m_meshPtr->vertT[i] = vertT_before_permute_h[i];
    }
    ans.push_back(m_meshPtr->vertT);
  }

  hipDeviceSynchronize();
  endtime = clock();
  if (verbose)
    printf("redistance time : %.10lf s\n", (LevelsetValueType)duration1 / CLOCKS_PER_SEC);
  if (verbose)
    printf("levelset update time : %.10lf s\n", (LevelsetValueType)duration2 / CLOCKS_PER_SEC);
  duration = (double)(endtime - starttime) / (double)CLOCKS_PER_SEC;
  if (verbose)
    printf("Processing time : %.10lf s\n", duration);
  return ans;
}

void meshFIM::getPartIndicesNegStart(IdxVector_d& sortedPartition, IdxVector_d& partIndices)
{
  // Sizing the array:
  int maxPart = sortedPartition[sortedPartition.size() - 1];
  partIndices.resize(maxPart + 2, 0);

  // Figuring out block sizes for kernel call:
  int size = sortedPartition.size();
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Getting pointers
  int *sortedPartition_d = thrust::raw_pointer_cast(&sortedPartition[0]);
  int *partIndices_d = thrust::raw_pointer_cast(&partIndices[0]);

  // Calling kernel to find indices for each part:
  findPartIndicesNegStartKernel << < nBlocks, blockSize >> > (size, sortedPartition_d, partIndices_d);
  partIndices[partIndices.size() - 1] = size - 1;
}

void meshFIM::mapAdjacencyToBlock(IdxVector_d &adjIndexes, IdxVector_d &adjacency, IdxVector_d &adjacencyBlockLabel, IdxVector_d &blockMappedAdjacency, IdxVector_d &fineAggregate)
{
  int size = adjIndexes.size() - 1;
  // Get pointers:adjacencyIn
  int *adjIndexes_d = thrust::raw_pointer_cast(&adjIndexes[0]);
  int *adjacency_d = thrust::raw_pointer_cast(&adjacency[0]);
  int *adjacencyBlockLabel_d = thrust::raw_pointer_cast(&adjacencyBlockLabel[0]);
  int *blockMappedAdjacency_d = thrust::raw_pointer_cast(&blockMappedAdjacency[0]);
  int *fineAggregate_d = thrust::raw_pointer_cast(&fineAggregate[0]);

  // Figuring out block sizes for kernel call:
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Calling kernel:
  mapAdjacencyToBlockKernel << < nBlocks, blockSize >> > (size, adjIndexes_d, adjacency_d, adjacencyBlockLabel_d, blockMappedAdjacency_d, fineAggregate_d);
}

