#include "hip/hip_runtime.h"
/*
Szymon Rusinkiewicz
Princeton University

TriMesh_connectivity.cc
Manipulate data structures that describe connectivity between faces and verts.
*/


#include <stdio.h>
#include "TriMesh.h"
#include <algorithm>
#include <stdlib.h>
#include <time.h>
#include <math.h>

using std::find;


void TriMesh::need_Rinscribe()
{
	need_faceedges();
	int nf = faces.size();
	if (!radiusInscribe.empty())
	{
		return;
	}

	radiusInscribe.resize(nf);
	
	for (int i=0; i<nf; i++)
	{
		Face f = faces[i];
		double e1 = f.edgeLens[0];
		double e2 = f.edgeLens[1];
		double e3 = f.edgeLens[2];
		double s = (e1+e2+e3)/2.0;
		radiusInscribe[i] = sqrt(s *(s-e1) * (s-e2) * (s - e3)) / s;
	}
	
}

void TriMesh::need_meshinfo()
{
	int nf = faces.size();
	int numObtuse = 0;
	float maxAngle = 0;
	for (int i =0; i< nf; i++)
	{
		Face f = faces[i];
		for (int j=0;j<3; j++)
		{
			point A = vertices[f[j]];
			point B = vertices[f[(j+1)%3]];
			point C = vertices[f[(j+2)%3]];
			point AB = B - A;
			point AC = C - A;
			float angle = acos( AB DOT AC / (sqrt(AB DOT AB) * sqrt(AC DOT AC)));
			maxAngle = MAX(maxAngle, angle);
			if (angle >= M_PI / 2.0)
			{
				numObtuse++;

			}
			

			
		}
		

	}

	printf("number of obtuse triangles/total triangles = %d / %d\n", numObtuse, nf);
	printf("maximum angles = %f\n", maxAngle * 180 / M_PI);
	
}

//compute the edge length
void TriMesh::need_faceedges()
{
	if (faces.empty())
	{
		printf("No faces to compute face edges!!!\n");
		return;
	}
	int numFaces = faces.size();
	for (int i = 0; i < numFaces; i++)
	{
		Face f = faces[i];
		point edge01 = vertices[f[1]] - vertices[f[0]];
		point edge12 = vertices[f[2]] - vertices[f[1]];
		point edge20 = vertices[f[0]] - vertices[f[2]];
		faces[i].edgeLens[0] =sqrt(edge01[0]*edge01[0] + edge01[1]*edge01[1] + edge01[2]*edge01[2]);
		faces[i].edgeLens[1] =sqrt(edge12[0]*edge12[0] + edge12[1]*edge12[1] + edge12[2]*edge12[2]);
		faces[i].edgeLens[2] =sqrt(edge20[0]*edge20[0] + edge20[1]*edge20[1] + edge20[2]*edge20[2]);


	}

}

vector<float> rescale(vector<float> scalar, float rangemin, float rangemax )
{
	float orimax = 0;
	float orimin = 100000000;
	int nv = scalar.size();
	vector<float> result;
	result.resize(nv);
	for (int i =0; i< nv; i++)
	{
		orimin = MIN(orimin, scalar[i]);
		orimax = MAX(orimax, scalar[i]);
	}

	for ( int i=0; i<nv; i++)
	{
		result[i] = ( rangemax*(scalar[i] - orimin) + rangemin*(orimax-scalar[i]) )/ (orimax -orimin) ;
	}

	return result;
	
	
}
void TriMesh::need_noise()
{
	if (!noiseOnVert.empty())
		return;


	FILE* file = fopen("noise_1.4m.txt","r");
	//FILE* file = fopen("noiseSphereR60_16k.txt","r");

	need_neighbors();
	int nv = vertices.size();
	int nf = faces.size();
	noiseOnVert.resize(nv);
	noiseOnFace.resize(nf);
	srand( (unsigned)time( NULL ) );

	for (int i = 0;i<nv; i++)
	{
		//float up = 2;
		//float down = 0;
		//noiseOnVert[i] = (float)rand() / (RAND_MAX + 1)*(up - down) + down;  //random number between [donw,up)
		float wgn; 
		fscanf(file,"%f",&wgn); 
		noiseOnVert[i] = wgn;


	}

	fclose(file);

	file = fopen("noise_1.4m.txt","r");

	for (int i = 0;i<nf; i++)
	{
		//float up = 2;
		//float down = 0;
		//noiseOnVert[i] = (float)rand() / (RAND_MAX + 1)*(up - down) + down;  //random number between [donw,up)
		float wgn; 
		fscanf(file,"%f",&wgn); 
		noiseOnFace[i] = wgn;


	}

	fclose(file);

	vector< float> tmpNoiseOnVert;
	tmpNoiseOnVert.resize(nv);

	float dt = 1.0 / 20.0;
	//iterate 
	int maxIterNum = 1;
	int iterStep = 10;

	//FILE* diffnoisefile = fopen("diffnoise.txt", "w+");

	//fprintf(diffnoisefile,"%d\n", nv);
	//fprintf(diffnoisefile,"%d\n", maxIterNum / iterStep + 1 );
	//for (int maxiternum = 0; maxiternum <= maxIterNum; maxiternum +=iterStep)
	//{


	//	for (int i=0;i<10/*maxiternum*/; i++)
	//	{
	//		for (int j=0; j<nf;j++)
	//		{

	//			//noiseOnVert[j] = 0;
	//			vector<int> nbs(across_edge[j][0],across_edge[j][1],across_edge[j][2]);
	//			float upvalue = 0;
	//			float downvalue = 0;

	//			for (int k=0;k<nbs.size();k++)
	//			{
	//				int nb = nbs[k];
	//				vector<int> nbnbs = neighbors[nb];
	//				vector<int> samenbs;
	//				//float areaPoly = 0;
	//				samenbs.clear();


	//				///////////////find the A and B//////////////////
	//				for (int numnbs =0; numnbs < nbnbs.size(); numnbs++)
	//				{
	//					for (int numCnbs = 0; numCnbs < nbs.size(); numCnbs++)
	//					{
	//						if (nbnbs[numnbs] == nbs[numCnbs])
	//						{
	//							samenbs.push_back(nbnbs[numnbs]);
	//						}

	//					}

	//				}

	//				//////////////////////////////////////////////////////
	//				if (samenbs.size() == 1)
	//				{
	//					point AC = vertices[j] - vertices[samenbs[0]];
	//					point AN = vertices[nb] - vertices[samenbs[0]];
	//					float alpha = acos( (AC DOT AN) / (sqrt(AC DOT AC) * sqrt( AN DOT AN)) );
	//					alpha = MIN(alpha,(float) 85.0*M_PI/180.0);
	//					alpha = MAX(alpha, (float)5.0*M_PI/180.0);
	//					float w = 1.0 / tan(alpha);
	//					downvalue += w;
	//					upvalue += w*noiseOnVert[nb];
	//					// upvalue +=0.5 * w* (noiseOnVert[j] - noiseOnVert[nb]);

	//				}
	//				else if (samenbs.size()==2)
	//				{
	//					point AC = vertices[j] - vertices[samenbs[0]];
	//					point AN = vertices[nb] - vertices[samenbs[0]];
	//					point BC = vertices[j] - vertices[samenbs[1]];
	//					point BN = vertices[nb] - vertices[samenbs[1]];
	//					float alpha = acos( (AC DOT AN) / (sqrt(AC DOT AC) * sqrt( AN DOT AN)) );
	//					alpha = MIN(alpha, (float)85.0*M_PI/180.0);
	//					alpha = MAX(alpha, (float)5.0*M_PI/180.0);

	//					float beta  = acos( (BC  DOT BN) / (sqrt(BC  DOT BC ) * sqrt( BN DOT BN)) );

	//					beta = MIN(beta, (float)85.0*M_PI/180.0);
	//					beta = MAX(beta,(float) 5.0*M_PI/180.0);
	//					float w = 1.0 / tan(alpha) + 1.0 / tan(beta);
	//					downvalue += w;
	//					upvalue += w*noiseOnVert[nb];
	//					//upvalue +=0.5 * w* (noiseOnVert[j] - noiseOnVert[nb]);

	//				}
	//				else
	//					printf("same nbs ERROR : not 1 or 2!!\n");



	//				//noiseOnVert[j] +=noiseOnVert[neighbors[j][k]];

	//			}
	//			//noiseOnVert[j] /= nb.size();


	//			float delta = noiseOnVert[j] - upvalue / downvalue;
	//			tmpNoiseOnVert[j] = noiseOnVert[j] - dt * delta;
	//		}

	//		for (int buzhidao = 0; buzhidao < nv; buzhidao++)
	//		{
	//			noiseOnVert[buzhidao] = tmpNoiseOnVert[buzhidao];
	//		}



	//	}
	//}



	for (int maxiternum = 0; maxiternum <= maxIterNum; maxiternum +=iterStep)
	{


		for (int i=0;i<20/*maxiternum*/; i++)
		{
			for (int j=0; j<nv;j++)
			{

				//noiseOnVert[j] = 0;
				vector<int> nbs = neighbors[j];
				float upvalue = 0;
				float downvalue = 0;

				for (int k=0;k<nbs.size();k++)
				{
					int nb = nbs[k];
					vector<int> nbnbs = neighbors[nb];
					vector<int> samenbs;
					//float areaPoly = 0;
					samenbs.clear();


					///////////////find the A and B//////////////////
					for (int numnbs =0; numnbs < nbnbs.size(); numnbs++)
					{
						for (int numCnbs = 0; numCnbs < nbs.size(); numCnbs++)
						{
							if (nbnbs[numnbs] == nbs[numCnbs])
							{
								samenbs.push_back(nbnbs[numnbs]);
							}

						}

					}

					//////////////////////////////////////////////////////
					if (samenbs.size() == 1)
					{
						point AC = vertices[j] - vertices[samenbs[0]];
						point AN = vertices[nb] - vertices[samenbs[0]];
						float alpha = acos( (AC DOT AN) / (sqrt(AC DOT AC) * sqrt( AN DOT AN)) );
						alpha = MIN(alpha,(float) 85.0*M_PI/180.0);
						alpha = MAX(alpha, (float)5.0*M_PI/180.0);
						float w = 1.0 / tan(alpha);
						downvalue += w;
						upvalue += w*noiseOnVert[nb];
						// upvalue +=0.5 * w* (noiseOnVert[j] - noiseOnVert[nb]);

					}
					else if (samenbs.size()==2)
					{
						point AC = vertices[j] - vertices[samenbs[0]];
						point AN = vertices[nb] - vertices[samenbs[0]];
						point BC = vertices[j] - vertices[samenbs[1]];
						point BN = vertices[nb] - vertices[samenbs[1]];
						float alpha = acos( (AC DOT AN) / (sqrt(AC DOT AC) * sqrt( AN DOT AN)) );
						alpha = MIN(alpha, (float)85.0*M_PI/180.0);
						alpha = MAX(alpha, (float)5.0*M_PI/180.0);

						float beta  = acos( (BC  DOT BN) / (sqrt(BC  DOT BC ) * sqrt( BN DOT BN)) );

						beta = MIN(beta, (float)85.0*M_PI/180.0);
						beta = MAX(beta,(float) 5.0*M_PI/180.0);
						float w = 1.0 / tan(alpha) + 1.0 / tan(beta);
						downvalue += w;
						upvalue += w*noiseOnVert[nb];
						//upvalue +=0.5 * w* (noiseOnVert[j] - noiseOnVert[nb]);

					}
					else
						printf("same nbs ERROR : not 1 or 2!!\n");



					//noiseOnVert[j] +=noiseOnVert[neighbors[j][k]];

				}
				//noiseOnVert[j] /= nb.size();


				float delta = noiseOnVert[j] - upvalue / downvalue;
				tmpNoiseOnVert[j] = noiseOnVert[j] - dt * delta;
			}

			for (int buzhidao = 0; buzhidao < nv; buzhidao++)
			{
				noiseOnVert[buzhidao] = tmpNoiseOnVert[buzhidao];
			}



		}

		//noiseOnVert = rescale(noiseOnVert, 0, 2);

	//	//fprintf(diffnoisefile,"%d\n", maxiternum);
	//	//for (int j=0; j<nv; j++)
	//	//{
	//	//	fprintf(diffnoisefile,"%f\n",noiseOnVert[j]);
	//	//}

	//	colors.resize(nv);
	//	for (int i =0; i< nv; i++)
	//	{
	//		colors[i] = Color(noiseOnVert[i]/2.0,noiseOnVert[i]/2.0,noiseOnVert[i]/2.0);
	//	}
	//	

	//	
	//	
	//	

	}

	//fclose(diffnoisefile);



}

void TriMesh::need_speed()
{
	int nf = faces.size();
	//need_noise();

	//FILE* file = fopen("noiseSphereR40.txt","r");
	//FILE* file = fopen("noiseSquare.1.txt","r");
	FILE* file = fopen("noise_1.4m.txt","r");
	
	for (int i =0; i<nf;i++)
	{
		Face f = faces[i];
		switch (SPEEDTYPE)
		{

			

		
			case CURVATURE:
				faces[i].speedInv = ( abs(curv1[f[0]] + curv2[f[0]]) + abs(curv1[f[1]] + curv2[f[1]]) + abs(curv1[f[2]] + curv2[f[2]]) )/ 6.0;
				break;
			case ONE:
				faces[i].speedInv = 1.0;
				break;
			case NOISE:
				faces[i].speedInv =( noiseOnVert[faces[i][0]] + noiseOnVert[faces[i][1]] + noiseOnVert[faces[i][2]] )/ 3;
				//float wgn; 
				//fscanf(file,"%f",&wgn); 
				//faces[i].speedInv = wgn;//( noiseOnVert[faces[i][0]] + noiseOnVert[faces[i][1]] + noiseOnVert[faces[i][2]] )/ 3;
				
				break;


		}

		
	}

	//fclose(file);
	
	

	
}


// Find the direct neighbors of each vertex
void TriMesh::need_neighbors(bool verbose)
{
	if (!neighbors.empty())
		return;
	need_faces();

	if (verbose)
    dprintf("Finding vertex neighbors... ");
	int nv = vertices.size(), nf = faces.size();

	vector<int> numneighbors(nv);
	for (int i = 0; i < nf; i++) {
		numneighbors[faces[i][0]]++;
		numneighbors[faces[i][1]]++;
		numneighbors[faces[i][2]]++;
	}

	neighbors.resize(nv);
	for (int i = 0; i < nv; i++)
		neighbors[i].reserve(numneighbors[i]+2); // Slop for boundaries

	for (int i = 0; i < nf; i++) {
		for (int j = 0; j < 3; j++) {
			vector<int> &me = neighbors[faces[i][j]];
			int n1 = faces[i][(j+1)%3];
			int n2 = faces[i][(j+2)%3];
			if (find(me.begin(), me.end(), n1) == me.end())
				me.push_back(n1);
			if (find(me.begin(), me.end(), n2) == me.end())
				me.push_back(n2);
		}
	}

  if (verbose)
	  dprintf("Done.\n");
}

void TriMesh::need_oneringfaces()
{

	if (vertOneringFaces.empty())
	{
		vertOneringFaces.resize(vertices.size());
		for (int i=0; i< vertices.size();i++)
		{
			vertOneringFaces[i] = GetOneRing(i);

		}
		
	}
	
}


// Find the faces touching each vertex
void TriMesh::need_adjacentfaces(bool verbose)
{
	if (!adjacentfaces.empty())
		return;
	need_faces();
  if (verbose)
	  dprintf("Finding vertex to triangle maps... ");
	int nv = vertices.size(), nf = faces.size();

	vector<int> numadjacentfaces(nv);
	for (int i = 0; i < nf; i++) {
		numadjacentfaces[faces[i][0]]++;
		numadjacentfaces[faces[i][1]]++;
		numadjacentfaces[faces[i][2]]++;
	}

	adjacentfaces.resize(vertices.size());
	for (int i = 0; i < nv; i++)
		adjacentfaces[i].reserve(numadjacentfaces[i]);

	for (int i = 0; i < nf; i++) {
		for (int j = 0; j < 3; j++)
			adjacentfaces[faces[i][j]].push_back(i);
	}

  if (verbose)
    dprintf("Done.\n");
}


// Find the face across each edge from each other face (-1 on boundary)
// If topology is bad, not necessarily what one would expect...
void TriMesh::need_across_edge()
{
	if (!across_edge.empty())
		return;
	need_adjacentfaces();

	dprintf("Finding across-edge maps... ");

	int nf = faces.size();
	across_edge.resize(nf, Face(-1,-1,-1));

	for (int i = 0; i < nf; i++) {
		for (int j = 0; j < 3; j++) {
			if (across_edge[i][j] != -1)
				continue;
			int v1 = faces[i][(j+1)%3];
			int v2 = faces[i][(j+2)%3];
			const vector<int> &a1 = adjacentfaces[v1];
			const vector<int> &a2 = adjacentfaces[v2];
			for (int k1 = 0; k1 < a1.size(); k1++) {
				int other = a1[k1];
				if (other == i)
					continue;
				vector<int>::const_iterator it =
					find(a2.begin(), a2.end(), other);
				if (it == a2.end())
					continue;
				int ind = (faces[other].indexof(v1)+1)%3;
				if (faces[other][(ind+1)%3] != v2)
					continue;
				across_edge[i][j] = other;
				across_edge[other][ind] = i;
				break;
			}
		}
	}

	dprintf("Done.\n");
}

