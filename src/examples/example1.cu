#include "hip/hip_runtime.h"
#include <LevelSet.h>
#include <cmath>

int main(int argc, char *argv[])
{
  LevelSet data(false,"../src/test/test_data/sphere334",false);
  std::string type = "x";
  //input filename (minus extension)
  for (int i = 0; i < argc; i++) {
    if (strcmp(argv[i],"-v") == 0) {
      data.verbose_ = true;
    } else if (strcmp(argv[i],"-i") == 0) {
      if (i+1 >= argc) break;
      data.filename_ = std::string(argv[i+1]);
      if (data.filename_.substr(data.filename_.size()-5,5) == ".node")
        data.filename_ = data.filename_.substr(0,data.filename_.size() - 5);
      if (data.filename_.substr(data.filename_.size()-4,4) == ".ele")
        data.filename_ = data.filename_.substr(0,data.filename_.size() - 4);
      i++;
    } else if (strcmp(argv[i],"-n") == 0) {
      if (i+1 >= argc) break;
      data.numSteps_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-t") == 0) {
      if (i+1 >= argc) break;
      data.timeStep_ = atof(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-s") == 0) {
      if (i+1 >= argc) break;
      data.insideIterations_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-d") == 0) {
      if (i+1 >= argc) break;
      data.sideLengths_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-p") == 0) {
      if (i+1 >= argc) break;
      data.partitionType_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-m") == 0) {
      if (i+1 >= argc) break;
      data.metisSize_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-b") == 0) {
      if (i+1 >= argc) break;
      data.blockSize_ = atoi(argv[i+1]);
      i++;
    } else if (strcmp(argv[i],"-w") == 0) {
      if (i+1 >= argc) break;
      data.bandwidth_ = atof(argv[i+1]);
      i++;
    } else if (strcmp(argv[i], "-y") == 0) {
      if (i + 1 >= argc) break;
      type = std::string(argv[++i]);
    } else if (strcmp(argv[i],"-h") == 0) {
      std::cout << "Usage: ./Example1 [OPTIONS]" << std::endl;
      std::cout << "   -h                 Print this help message." << std::endl;
      std::cout << "   -v                 Print verbose runtime information." << std::endl;
      std::cout << "   -i FILENAME        Use this input tet mesh (node/ele)." << std::endl;
      std::cout << "   -n NSTEPS          # of steps to take of TIMESTEP amount." << std::endl;
      std::cout << "   -t TIMESTEP        Duration of a timestep." << std::endl;
      std::cout << "   -s INSIDE_NITER    # of inside iterations." << std::endl;
      std::cout << "   -d NSIDE           # of sides for Square partition type." << std::endl;
      std::cout << "   -p PARTITION_TYPE  1 for Square, otherwise is it METIS." << std::endl;
      std::cout << "   -b NUM_BLOCKS      # of blocks for Square partition type." << std::endl;
      std::cout << "   -m METIS_SIZE      The size for METIS partiation type." << std::endl;
      std::cout << "   -w BANDWIDTH       The Bandwidth for the algorithm." << std::endl;
      std::cout << "   -y EXAMPLE_TYPE    Example type: 'center', 'revolve', 'x'" << std::endl;
      exit(0);
    }
  }
  if (type == "center"  || type == "revolve") {
    //find the center, max from center
    data.initializeMesh();
    point center(0, 0, 0);
    for (size_t i = 0; i < data.tetMesh_->vertices.size(); i++) {
      center = center + data.tetMesh_->vertices[i];
    }
    center = center / static_cast<float>(data.tetMesh_->vertices.size());
    float max = 0.;
    for (size_t i = 0; i < data.tetMesh_->vertices.size(); i++) {
      point p = data.tetMesh_->vertices[i] - center;
      float mag = len(p);
      max = std::max(max, mag);
    }
    //initialize values of verts
    std::vector<float> vals;
    for (size_t i = 0; i < data.tetMesh_->vertices.size(); i++) {
      point p = data.tetMesh_->vertices[i] - center;
      double mag = len(p);
      if (type == "revolve") {
        //get the angle with (+/-1,0,0)
        float val = p[0];
        if (val < 0.) val *= -1.;
        float theta = std::acos(val / std::sqrt(p[0] * p[0] + p[1] * p[1]));
        if (p[1] < 0.f) theta *= -1.f;
        vals.push_back(10.f * theta);
      } else {
        vals.push_back(mag - max / 2.);
      }
    }
    //initialize advection to be away from the center.
    std::vector<point> adv;
    for (size_t i = 0; i < data.tetMesh_->tets.size(); i++) {
      point p = (data.tetMesh_->vertices[data.tetMesh_->tets[i][0]] +
        data.tetMesh_->vertices[data.tetMesh_->tets[i][1]] +
        data.tetMesh_->vertices[data.tetMesh_->tets[i][2]] +
        data.tetMesh_->vertices[data.tetMesh_->tets[i][3]])
        / 4.f - center;
      float mag = len(p);
      mag /= max / 20.f;
      if (type == "revolve") {
        //only care about XY plane angle
        //get the tangent to the central circle
        point p2 = p;
        p2[2] = 0.f;
        point p3 = p2 CROSS point(0, 0, 1);
        adv.push_back(p3 * len(p2) / (100.f * len (p3)));
      } else {
        adv.push_back(p / mag / mag);
      }
    }
    data.initializeVertices(vals);
    data.initializeAdvection(adv);
  }
  data.solveLevelSet();
  data.writeVTK();
  return 0;
}

